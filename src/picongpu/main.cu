#include "hip/hip_runtime.h"
/**
 * Copyright 2013 Axel Huebl, Felix Schmitt, Heiko Burau, Rene Widera
 *
 * This file is part of PIConGPU.
 *
 * PIConGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * PIConGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with PIConGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * @mainpage PIConGPU-Frame
 *
 * Project with HZDR for porting their PiC-code to a GPU cluster.
 *
 * \image html picongpu.jpg
 *
 * @author Heiko Burau, Rene Widera, Wolfgang Hoenig, Felix Schmitt, Axel Huebl, Michael Bussmann, Guido Juckeland
 */


// include the heap with the arguments given in the config
#include "mallocMC/mallocMC_utils.hpp"

// basic files for mallocMC
#include "mallocMC/mallocMC_overwrites.hpp"
#include "mallocMC/mallocMC_hostclass.hpp"

// Load all available policies for mallocMC
#include "mallocMC/CreationPolicies.hpp"
#include "mallocMC/DistributionPolicies.hpp"
#include "mallocMC/OOMPolicies.hpp"
#include "mallocMC/ReservePoolPolicies.hpp"
#include "mallocMC/AlignmentPolicies.hpp"

// configurate the CreationPolicy "Scatter"
struct ScatterConfig
{
    /* 2MiB page can hold around 256 particle frames */
    typedef boost::mpl::int_<2*1024*1024> pagesize;
    /* accessblocks, regionsize and wastefactor are not finale selected
       and might be performance sensitive*/
    typedef boost::mpl::int_<4> accessblocks;
    typedef boost::mpl::int_<8> regionsize;
    typedef boost::mpl::int_<2> wastefactor;
    /* resetfreedpages is used to minimize memory fracmentation while different
       frame sizes were used*/
    typedef boost::mpl::bool_<true> resetfreedpages;
};

// Define a new allocator and call it ScatterAllocator
// which resembles the behaviour of ScatterAlloc
typedef mallocMC::Allocator<
mallocMC::CreationPolicies::Scatter<ScatterConfig>,
mallocMC::DistributionPolicies::Noop,
mallocMC::OOMPolicies::ReturnNull,
mallocMC::ReservePoolPolicies::SimpleCudaMalloc,
mallocMC::AlignmentPolicies::Shrink<>
> ScatterAllocator;

//use ScatterAllocator to replace malloc/free
MALLOCMC_SET_ALLOCATOR_TYPE( ScatterAllocator );


#include "debug/CrashDump.hpp"
#include <simulation_defines.hpp>

#include "communication/manager_common.h"
#include "debug/LogStatus.hpp"
#include "Environment.hpp"

#include <mpi.h>
#include <exception>
#include <sstream>

using namespace PMacc;
using namespace picongpu;


namespace picongpu
{

template<typename T_Type>
void writeCrashDump( const T_Type& simulation )
{
    std::stringstream debugOutput;
    debugOutput << debug::logStatus( simulation, "sim" ) << "\n";
    debugOutput<<"-----------------------Transactions---------------"<<"\n";
    debugOutput << debug::logStatus( PMacc::Environment<>::get( ).TransactionManager( ) ) << "\n";
    debugOutput<<"-----------------------Manager---------------"<<"\n";
    debugOutput << debug::logStatus( PMacc::Environment<>::get( ).Manager( ) ) << "\n";
    std::cerr << "create crash log" << std::endl;
    PMacc::debug::CrashDump::getInstance().dumpToFile( debugOutput.str( ), "error" );
}

} //picongpu

/*! start of PIConGPU
 *
 * @param argc count of arguments in argv
 * @param argv arguments of program start
 */
int main( int argc, char **argv )
{
    MPI_CHECK( MPI_Init( &argc, &argv ) );
    PMacc::debug::CrashDump::getInstance().init();

    picongpu::simulation_starter::SimStarter sim;
    try
    {

        if ( !sim.parseConfigs( argc, argv ) )
        {
            MPI_CHECK( MPI_Finalize( ) );
            return 1;
        }

        sim.load( );
        sim.start( );
        sim.unload( );

        MPI_CHECK( MPI_Finalize( ) );
    }
    catch ( std::logic_error& e )
    {
        std::cerr << e.what( ) << std::endl;
        picongpu::writeCrashDump( sim );
        return 1;
    }
    catch ( std::runtime_error& e )
    {
        std::cerr << e.what( ) << std::endl;
        picongpu::writeCrashDump( sim );
        return 1;
    }
    catch ( std::exception& e )
    {
        std::cerr << e.what( ) << std::endl;
        picongpu::writeCrashDump( sim );
        return 1;
    }

    return 0;
}
